#include "hip/hip_runtime.h"
#include <cstdint>

#include "Cuda.hpp"
#include "Util_Cuda.hpp"

namespace cbm {

  template<typename ScaType>
  __global__ void set_to_zero(ScaType* p, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] = 0;
  }

  template<typename ScaType>
  __global__ void set_to_one(ScaType* p, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] = 1;
  }

  template<typename ScaType>
  __global__ void set_to_const(ScaType* p, ScaType v, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] = v;
  }

  template<typename ScaType>
  __global__ void inplace_multiply(ScaType* p, ScaType v, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] *= v;
  }

  template<typename ScaType>
  __global__ void inplace_divide(ScaType* p, ScaType v, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] /= v;
  }

  template<typename ScaType>
  __global__ void inplace_add(ScaType* p, ScaType v, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] += v;
  }

  template<typename ScaType>
  __global__ void inplace_minus(ScaType* p, ScaType v, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) p[idx] -= v;
  }

  // set increasing values to an array
  // set an array to v0, v0+inc, v0+2*inc,..., v0+(len-1)*inc
  template<typename ScaType>
  __global__ void inplace_set_inc(ScaType* p, ScaType v0, ScaType inc, int stride, int len) {
    int idx = LINEAR_IDX;
    if (idx < len) {
      int i = idx*stride;
      p[i] = v0 + idx*inc;
    }
  }
  
  template __global__ void set_to_zero<float>(float* p, int len);
  template __global__ void set_to_zero<double>(double* p, int len);
  template __global__ void set_to_zero<int>(int* p, int len);

  template __global__ void set_to_one<float>(float* p, int len);
  template __global__ void set_to_one<double>(double* p, int len);
  template __global__ void set_to_one<int>(int* p, int len);

  template __global__ void set_to_const<float>(float* p, float v, int len);
  template __global__ void set_to_const<double>(double* p, double v, int len);
  template __global__ void set_to_const<int>(int* p, int v, int len);


  template __global__ void inplace_multiply<float>(float* p, float v, int len);
  template __global__ void inplace_multiply<double>(double* p, double v, int len);
  template __global__ void inplace_multiply<int>(int* p, int v, int len);

  template __global__ void inplace_divide<float>(float* p, float v, int len);
  template __global__ void inplace_divide<double>(double* p, double v, int len);
  template __global__ void inplace_divide<int>(int* p, int v, int len);

  template __global__ void inplace_add<float>(float* p, float v, int len);
  template __global__ void inplace_add<double>(double* p, double v, int len);
  template __global__ void inplace_add<int>(int* p, int v, int len);

  template __global__ void inplace_minus<float>(float* p, float v, int len);
  template __global__ void inplace_minus<double>(double* p, double v, int len);
  template __global__ void inplace_minus<int>(int* p, int v, int len);

  template __global__ void inplace_set_inc<float>(float* p, float v0, float inc, int stride, int len);
  template __global__ void inplace_set_inc<double>(double* p, double v0, double inc, int stride, int len);
  template __global__ void inplace_set_inc<int>(int* p, int v0, int inc, int stride, int len);
  template __global__ void inplace_set_inc<std::intptr_t>(std::intptr_t* p, std::intptr_t v0, std::intptr_t inc, int stride, int len);
  
}  // namespace cbm