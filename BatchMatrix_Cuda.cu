#include "hip/hip_runtime.h"
// Include CUDA implementations of BatchMatrix

#include <cstdint>
#include <memory>

#include <hip/hip_runtime.h>

#include "Log.hpp"
#include "Cuda.hpp"
#include "Util_Cuda.hpp"
#include "BatchMatrix.hpp"

namespace cbm {
  // Constructors
  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>::BatchMatrix(const BatchMatrix<ScaType, MemType>& t)
    : ptr_(nullptr) {
    static_assert(MemType==CUDA, "This function is only for constructing CUDA from CUDA");
    CUDA_CHECK(hipMalloc(&data_, t.len()*sizeof(ScaType)));
    CUDA_CHECK(hipMemcpy( data_, t.data(), t.len()*sizeof(ScaType), hipMemcpyDeviceToDevice));

    for (int i=0; i<3; i++) {
      dim_[i]    = t.dim()[i];
      stride_[i] = t.stride()[i];
    }

    update_ptr();
  }

  // Destructor
  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>::~BatchMatrix() {
    static_assert(MemType==CUDA, "This function is only for CUDA");

    if (data_!=nullptr) CUDA_CHECK(hipFree(data_));
  }

  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType,MemType>::BatchMatrix(int a, int b, int c) : ptr_(nullptr) {
    static_assert(MemType==CUDA, "This function is only for CUDA");

    // copy
    dim_[0] = a; dim_[1] = b; dim_[2] = c;
    
    // default: column major for each matrix
    stride_[1] = 1;
    stride_[2] = dim_[1];
    stride_[0] = dim_[1]*dim_[2];
    len_       = stride_[0]*dim_[0];
    
    CUDA_CHECK(hipMalloc(&data_, len_*sizeof(ScaType)));

    update_ptr();
  }


  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType,MemType>::BatchMatrix(const std::vector<int>& d)
    : BatchMatrix(d[0], d[1], d[2]) {}

  template<typename ScaType, Type MemType>
  void BatchMatrix<ScaType,MemType>::update_ptr() {
    if (ptr_!=nullptr) CUDA_CHECK(hipFree(ptr_));
    
    CUDA_CHECK(hipMalloc(&ptr_, dim_[0]*sizeof(ScaType*)));

    std::unique_ptr<ScaType*[]> tmp(new ScaType*[dim_[0]]);
    for (int i = 0; i < dim_[0]; i++) tmp[i] = data_ + i*stride_[0];
    CUDA_CHECK(hipMemcpy(ptr_, tmp.get(), dim_[0]*sizeof(ScaType*), hipMemcpyHostToDevice));

    // //!The following code also works, but calling a kernel instead!
    //    inplace_set_inc<<<std::floor(dim_[0]/TPB)+1, TPB>>>(reinterpret_cast<std::intptr_t*>(ptr_),
    //    							reinterpret_cast<std::intptr_t>(data_),
    //    							static_cast<std::intptr_t>(stride_[0]*sizeof(std::intptr_t)),
    //       							1, dim_[0]);
         
  }

  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>*
    BatchMatrix<ScaType, MemType>::ones(const std::vector<int>& d) {
    static_assert(MemType==CUDA, "This function is only for CUDA");
    auto ret = new BatchMatrix<ScaType, MemType>(d);
    set_to_const<<<std::floor(ret->len()/TPB)+1, TPB>>>(ret->data(), static_cast<ScaType>(1), ret->len());

    CUDA_CHECK(hipDeviceSynchronize());
    
    return ret;
  }

  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>*
    BatchMatrix<ScaType, MemType>::zeros(const std::vector<int>& d) {
    static_assert(MemType==CUDA, "This function is only for CUDA");
    auto ret = new BatchMatrix<ScaType, MemType>(d);
    set_to_const<<<std::floor(ret->len()/TPB)+1, TPB>>>(ret->data(), static_cast<ScaType>(0), ret->len());
    return ret;
  }

  template<typename ScaType, Type MemType>
  void BatchMatrix<ScaType, MemType>::add(ScaType v) {
    inplace_add<<<std::floor(len_/TPB)+1, TPB>>>(data_, v, len_);
  }

  template class BatchMatrix<float, CUDA>;
  template class BatchMatrix<int, CUDA>;
  template class BatchMatrix<double, CUDA>;
    
    
}  // namespace cbm
