#include "hip/hip_runtime.h"
// Include CUDA implementations of BatchMatrix

#include <hip/hip_runtime.h>

#include "Cuda.hpp"
#include "Util_Cuda.hpp"
#include "BatchMatrix.hpp"

namespace cbm {
  // Constructors
  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>::BatchMatrix(const BatchMatrix<ScaType, MemType>& t) {
    static_assert(MemType==CUDA, "This function is only for constructing CUDA from CUDA");
    CUDA_CHECK(hipMalloc(&data_, t.len()*sizeof(ScaType)));
    CUDA_CHECK(hipMemcpy( data_, t.data(), t.len()*sizeof(ScaType), hipMemcpyDeviceToDevice));

    for (int i=0; i<3; i++) {
      dim_[i]    = t.dim()[i];
      stride_[i] = t.stride()[i];
    }
  }

  // Destructor
  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>::~BatchMatrix() {
    static_assert(MemType==CUDA, "This function is only for CUDA");

    if (data_!=nullptr) CUDA_CHECK(hipFree(data_));
  }


  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType,MemType>::BatchMatrix(int a, int b, int c) {
    static_assert(MemType==CUDA, "This function is only for CUDA");

    // copy
    dim_[0] = a; dim_[1] = b; dim_[2] = c;
    
    // default: column major for each matrix
    stride_[1] = 1;
    stride_[2] = dim_[1];
    stride_[0] = dim_[1]*dim_[2];
    len_       = stride_[0]*dim_[0];

    CUDA_CHECK(hipMalloc(&data_, len_*sizeof(ScaType)));
  }


  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType,MemType>::BatchMatrix(const std::vector<int>& d)
    : BatchMatrix(d[0], d[1], d[2]) {}


  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>*
    BatchMatrix<ScaType, MemType>::ones(const std::vector<int>& d) {
    static_assert(MemType==CUDA, "This function is only for CUDA");
    auto ret = new BatchMatrix<ScaType, MemType>(d);
    set_to_const<<<std::floor(ret->len()/TPB)+1, TPB>>>(ret->data(), static_cast<ScaType>(1), ret->len());
    return ret;
  }

  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>*
    BatchMatrix<ScaType, MemType>::zeros(const std::vector<int>& d) {
    static_assert(MemType==CUDA, "This function is only for CUDA");
    auto ret = new BatchMatrix<ScaType, MemType>(d);
    set_to_const<<<std::floor(ret->len()/TPB)+1, TPB>>>(ret->data(), static_cast<ScaType>(0), ret->len());
    return ret;
  }

  // Clone (CUDA to CUDA)
  template<typename ScaType, Type MemType>
  BatchMatrix<ScaType, MemType>* BatchMatrix<ScaType, MemType>::clone_cuda() const {
    static_assert(MemType==CUDA, "This function is only for CUDA to CUDA clone");
    auto ret = new BatchMatrix<ScaType, MemType>(dim_[0], dim_[1], dim_[2]);
    CUDA_CHECK(hipMemcpy(ret->data(), data_, len_*sizeof(MemType),hipMemcpyDeviceToDevice));
    return ret;
  };


  
  template<typename ScaType, Type MemType>
  void BatchMatrix<ScaType, MemType>::add(ScaType v) {
    inplace_add<<<std::floor(len_/TPB)+1, TPB>>>(data_, v, len_);
  }

  template class BatchMatrix<float, CUDA>;
  template class BatchMatrix<int, CUDA>;
  template class BatchMatrix<double, CUDA>;
    
    
}  // namespace cbm
