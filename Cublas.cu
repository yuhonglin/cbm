#include <iostream>
#include <mutex>

#include "Cublas.hpp"

namespace cbm {
  namespace cublas {

    // the default cublas handle
    hipblasHandle_t default_cublas_handle;

    static const char *cublasGetErrorString(hipblasStatus_t error)
    {
      switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
	  return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
	  return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
	  return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
	  return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
	  return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
	  return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
	  return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
	  return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

      return "<unknown>";
    }
    
    template<>
    void cublasGetrfBatched<float>(hipblasHandle_t& handle, int n, float *Array[],
				   int lda, int *PivotArray, int*infoArray,
				   int batchSize) {
      CUBLAS_CHECK(hipblasSgetrfBatched(handle, n, Array, lda, PivotArray, infoArray, batchSize));
    };


    template<>
    void cublasGetrfBatched<double>(hipblasHandle_t& handle, int n, double *Array[],
				    int lda, int *PivotArray, int*infoArray,
				    int batchSize) {
      CUBLAS_CHECK(hipblasDgetrfBatched(handle, n, Array, lda, PivotArray, infoArray, batchSize));
    };


    template<> void cublasGemmBatched<float>(hipblasHandle_t& handle, hipblasOperation_t transa,
					     hipblasOperation_t transb, int m, int n, int k,
					     float alpha, float *Aarray[], int lda, float *Barray[], int ldb,
					     float beta, float *Carray[], int ldc, int batchCount) {
      CUBLAS_CHECK(hipblasSgemmBatched(handle, transa, transb, m, n, k, &alpha, const_cast<const float**>(Aarray), lda,
				      const_cast<const float**>(Barray), ldb, &beta, Carray, ldc, batchCount));
    }

    template<> void cublasGemmBatched<double>(hipblasHandle_t& handle, hipblasOperation_t transa,
					     hipblasOperation_t transb, int m, int n, int k,
					     double alpha, double *Aarray[], int lda, double *Barray[], int ldb,
					     double beta, double *Carray[], int ldc, int batchCount) {
      CUBLAS_CHECK(hipblasDgemmBatched(handle, transa, transb, m, n, k, &alpha, const_cast<const double**>(Aarray), lda,
				      const_cast<const double**>(Barray), ldb, &beta, Carray, ldc, batchCount));
    }

    
    void init() {
      static bool first = true;
      if (first) {
	CUBLAS_CHECK(hipblasCreate(&default_cublas_handle));
	first = false;
      }
    }

    void clear() {
      hipblasDestroy(default_cublas_handle);
    }
  }
}
