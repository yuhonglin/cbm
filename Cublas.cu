#include <iostream>
#include <mutex>

#include "Cublas.hpp"

namespace cbm {
  namespace cublas {

    // the default cublas handle
    hipblasHandle_t default_cublas_handle;

    static const char *cublasGetErrorString(hipblasStatus_t error)
    {
      switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
	  return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
	  return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
	  return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
	  return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
	  return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
	  return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
	  return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
	  return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

      return "<unknown>";
    }
    
    template<>
    void cublasGetrfBatched<float>(hipblasHandle_t& handle, int n, float *Array[],
				   int lda, int *PivotArray, int*infoArray,
				   int batchSize) {
      CUBLAS_CHECK(hipblasSgetrfBatched(handle, n, Array, lda, PivotArray, infoArray, batchSize));
    };


    template<>
    void cublasGetrfBatched<double>(hipblasHandle_t& handle, int n, double *Array[],
				    int lda, int *PivotArray, int*infoArray,
				    int batchSize) {
      CUBLAS_CHECK(hipblasDgetrfBatched(handle, n, Array, lda, PivotArray, infoArray, batchSize));
    };


    void init() {
      static bool first = true;
      if (first) {
	CUBLAS_CHECK(hipblasCreate(&default_cublas_handle));
	first = false;
      }
    }

    void clear() {
      hipblasDestroy(default_cublas_handle);
    }
  }
}
